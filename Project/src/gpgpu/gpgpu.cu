#include "hip/hip_runtime.h"
#include <gpgpu.h>
#include <algorithm>
#include <iostream>
#include <random>

__device__ float2 operator-(float2 a, float2 b) {
	return make_float2(a.x - b.x, a.y - b.y);
};

void GetGPGPUInfo() {
	hipDeviceProp_t cuda_propeties;
	hipGetDeviceProperties(&cuda_propeties, 0);
	std::cout << "maxThreadsPerBlock: " << cuda_propeties.maxThreadsPerBlock << std::endl;
}

void Init(Fox** f_i, int nf_i, Rabbit** r_i, int nr_i) {
	hipMalloc((void**) f_i, sizeof(Fox) * nf_i);
	hipMalloc((void**) r_i, sizeof(Fox) * nr_i);
}

__global__ void kernel_uv(hipSurfaceObject_t surface, int32_t width, int32_t height, float time) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	float u = (float)x / width;
	float v = (float)y / height;
	float4 color = make_float4(u, v, cos(time), 1.0f);
	surf2Dwrite(color, surface, x * sizeof(float4), y);
}


__device__ void kernel_draw_rabbit(hipSurfaceObject_t surface, int x, int y, int32_t width, int32_t height, Rabbit* rabbits, int32_t nb_rabbits) {
	float4 RABBIT_COLOR = make_float4(1.f, 1.f, 1.f, 1.0f);
	float2 uv;
	uv.x = (float)x / width;
	uv.y = (float)y / height;

	for (int n = 0; n < nb_rabbits; ++n) {
		if (rabbits[n].is_alive) {
			if (hypotf(rabbits[n].u - uv.x, rabbits[n].v - uv.y) < rabbits[n].radius) {
				surf2Dwrite(RABBIT_COLOR, surface, x * sizeof(float4), y);
			}
		}
	}
}

__device__ void kernel_draw_fox(hipSurfaceObject_t surface, int x, int y, int32_t width, int32_t height, Fox* foxs, int32_t nb_foxs) {

	
	float2 uv;
	uv.x = (float)x / width;
	uv.y = (float)y / height;

	for (int n = 0; n < nb_foxs; ++n) {
		if (foxs[n].is_alive) {
			if (hypotf(foxs[n].u - uv.x, foxs[n].v - uv.y) < foxs[n].radius) {
				float degrade = 1.f - foxs[n].starvation / 50;
				float4 FOX_COLOR = make_float4(degrade, 0.f, 0.f, 1.0f);
				surf2Dwrite(FOX_COLOR, surface, x * sizeof(float4), y);
			}
		}
	}
}

__global__  void kernel_draw_map(hipSurfaceObject_t surface, int32_t width, int32_t height, Fox* foxs, int32_t nb_foxs, Rabbit* rabbits, int32_t nb_rabbits) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	float4 color = make_float4(0.6f, 0.9f, 0.05f, 1.0f);

	surf2Dwrite(color, surface, x * sizeof(float4), y);
	kernel_draw_fox(surface, x, y, width, height, foxs, nb_foxs);
	kernel_draw_rabbit(surface, x, y, width, height, rabbits, nb_rabbits);
}

__global__ void kernel_copy(hipSurfaceObject_t surface_in, hipSurfaceObject_t surface_out) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	float4 color = make_float4(1.f, 0.f, 1.f, 1.0f);
	surf2Dread(&color, surface_in, x * sizeof(float4), y);
	surf2Dwrite(color, surface_out, x * sizeof(float4), y);
}

__device__ float fracf(float x)
{
	return x - floorf(x);
}

__device__ float random(float x, float y) {
	float t = 12.9898f * x + 78.233f * y;
	return abs(fracf(t * sin(t)));
}

__global__ void moveFoxs(Fox* fox, int32_t nb_foxs, Rabbit* rabbits, int32_t nb_rabbits) {
	int index = threadIdx.x; // 0 - nbFoxes
	if (fox[index].is_alive) {
		float angle = random(fox[index].u, fox[index].v) * 2.f - 1.f;
		float modifier = 0.5f;
		float temp = cos(angle * modifier) * fox[index].direction_u + sin(angle * modifier) * fox[index].direction_v;
		fox[index].direction_v = -fox[index].direction_u * sin(angle * modifier) + cos(angle * modifier) * fox[index].direction_v;
		fox[index].direction_u = temp;

		for (int i = 0; i < nb_rabbits; ++i) {
			if (rabbits[i].is_alive) {
				if (hypotf(fox[index].u - rabbits[i].u, fox[index].v - rabbits[i].v) < fox[index].detection_radius) {
					fox[index].direction_u = (rabbits[i].u - fox[index].u);
					fox[index].direction_v = (rabbits[i].v - fox[index].v);
				}
				if (hypotf(fox[index].u - rabbits[i].u, fox[index].v - rabbits[i].v) < fox[index].radius) {
					fox[index].starvation_modifier += 0.005f;
					fox[index].radius += 0.0002f;
					fox[index].detection_radius += 0.0006f;
					fox[index].eatenPrey++;
					rabbits[i].is_alive = false;
					fox[index].starvation -= 10.f;
				}
				//break;
			}
		}

		float norm = sqrt(pow(fox[index].direction_u, 2) + pow(fox[index].direction_v, 2));
		fox[index].direction_u = fox[index].direction_u / norm;
		fox[index].direction_v = fox[index].direction_v / norm;

		float new_pos_x = fox[index].u + fox[index].direction_u * fox[index].speed;
		float new_pos_y = fox[index].v + fox[index].direction_v * fox[index].speed;

		if ((new_pos_x + fox[index].radius < 1) && (new_pos_x - fox[index].radius > 0))
			fox[index].u = new_pos_x;
		else
			fox[index].direction_u = -fox[index].direction_u;
		if ((new_pos_y + fox[index].radius < 1) && (new_pos_y - fox[index].radius > 0))
			fox[index].v = new_pos_y;
		else
			fox[index].direction_v = -fox[index].direction_v;

		fox[index].starvation += 0.015f + fox[index].starvation_modifier;
		if (fox[index].starvation >= fox[index].max_starvation) {
			fox[index].is_alive = false;
		}

		if (fox[index].eatenPrey >= fox[index].max_eatenPrey) {
			int i = 0;
			while (fox[i].is_alive) {
				++i;
			}

			if (i != nb_foxs) {
				fox[index].eatenPrey = 0;
				fox[i].u = fox[index].u;
				fox[i].v = fox[index].v;
				fox[i].is_alive = true;
				fox[i].starvation = 0.f;
				fox[i].eatenPrey = 0;
				fox[i].starvation_modifier = 0.f;
				fox[i].radius = 0.01f;
				fox[i].detection_radius = 1.f / 30.f;
			}
		}
	}
}

__global__ void moveRabbits(Rabbit* rabbit, int32_t nb_rabbits) {
	int index = threadIdx.x;
	if (rabbit[index].is_alive) {
		float angle = random(rabbit[index].u, rabbit[index].v) * 2.f - 1.f;
		float modifier = 0.3f;

		float temp = cos(angle * modifier) * rabbit[index].direction_u + sin(angle * modifier) * rabbit[index].direction_v;
		rabbit[index].direction_v = -rabbit[index].direction_u * sin(angle * modifier) + cos(angle * modifier) * rabbit[index].direction_v;
		rabbit[index].direction_u = temp;

		float new_pos_x = rabbit[index].u + rabbit[index].direction_u * rabbit[index].speed;
		float new_pos_y = rabbit[index].v + rabbit[index].direction_v * rabbit[index].speed;

		if ((new_pos_x + rabbit[index].radius < 1) && (new_pos_x - rabbit[index].radius > 0))
			rabbit[index].u = new_pos_x;
		else
			rabbit[index].direction_u = - rabbit[index].direction_u;
		if ((new_pos_y + rabbit[index].radius < 1) && (new_pos_y - rabbit[index].radius > 0))
			rabbit[index].v = new_pos_y;
		else
			rabbit[index].direction_v = - rabbit[index].direction_v;

		if (abs(angle) < 0.0008f) {
			int i = 0;
			while (rabbit[i].is_alive){
				++i;
			}

			if (i != nb_rabbits){
				rabbit[i].is_alive = true;
				rabbit[i].u = rabbit[index].u;
				rabbit[i].v = rabbit[index].v;
			}
		}
	}
}

void DrawUVs(hipSurfaceObject_t surface, int32_t width, int32_t height, float time) {
	dim3 threads(32, 32);
	dim3 blocks(32, 32);
	kernel_uv << <blocks, threads >> > (surface, width, height, time);
}

void DrawFoxs(Fox* foxs, int32_t nb_foxs, Rabbit* rabbits, int32_t nb_rabbits) {
	moveFoxs << < 1, nb_foxs >> > (foxs, nb_foxs, rabbits, nb_rabbits);
}

void DrawRabbits(Rabbit* rabbits, int32_t nb_rabbits) {
	moveRabbits << < 1, nb_rabbits >> > (rabbits, nb_rabbits);
}

void DrawMap(hipSurfaceObject_t surface, int32_t width, int32_t height, Fox* foxs, int32_t nb_foxs, Rabbit* rabbits, int32_t nb_rabbits) {
	dim3 threads(32, 32);
	dim3 blocks(32, 32);

	kernel_draw_map << <blocks, threads >> > (surface, width, height, foxs, nb_foxs, rabbits, nb_rabbits);
	/*
	for (int i = 0; i < width; ++i) {
		for (int j = 0; j < height; ++j) {
			
			//kernel_draw_fox << <32 * 3, 1024 >> > (surface, i, j, width, height, foxs, nb_foxs);
		}
	}*/
}

void CopyTo(hipSurfaceObject_t surface_in, hipSurfaceObject_t surface_out, int32_t width, int32_t height) {
	dim3 threads(32, 32);
	dim3 blocks(32, 32);
	kernel_copy << <blocks, threads >> > (surface_in, surface_out);
}